
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#define N 4;

//cudaError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

//__global__ void addKernel(int *c, const int *a, const int *b)
//{
//    int i = threadIdx.x;
//    c[i] = a[i] + b[i];
//}

__global__ void addKernel(double *a, double *b, double *c,int size)
{
    int i = threadIdx.x;
	for(int k = 0;k<size; k++)
	{
		if(i < size)
			c[i] += a[i*size+k] * b[k];
	}
}

void simple_dgemv(double *A, double *B, double *C,int size)
{
	int i,j;

	for(i = 0;i < size; i++)
	{
		double prod = 0;

		for(j = 0;j < size; j++)
		{	
			prod += A[i * size + j] * B[j];
		}
		C[i] = prod;
	}
}

//__global__ void MVKernel_gm(double *A, double *X, double *Y,int ARRAY_SIZE)
//{
//	//int bx = blockIdx.x; 
//          //int by = blockIdx.y;
//	int tid = threadIdx.x; 
//          //int ty = threadIdx.y;
//	// Calculate the row index of the Pd element and M
//	//int Row = bx * BLOCK_SIZE + tx;
//	// Calculate the column idenx of Pd and N
//	//int Col = bx * BLOCK_SIZE + tx;
//  
//	double tmpSum = 0;
//
//	for (int k = 0; k < ARRAY_SIZE; k++) 
//    {
//      if(tid < ARRAY_SIZE)
//      tmpSum += A[tid*ARRAY_SIZE+k] * X[k];
//    }
//
//	__syncthreads();
//  
//	if(tid < ARRAY_SIZE)  		
//		Y[tid] = tmpSum;
//
//	__syncthreads();
//}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

	int ARRAY_SIZE = 5;
	int ARRAY_SIZE2 = ARRAY_SIZE*ARRAY_SIZE;

	//Host
	double *h_a;
	double *h_b;
	double *h_c;

	//Device
	double *d_a;
	double *d_b;
	double *d_c;

	//generate the input array on the host
	h_a=(double*)malloc(sizeof(double)*ARRAY_SIZE2);
    h_b=(double*)malloc(sizeof(double)*ARRAY_SIZE);
    h_c=(double*)malloc(sizeof(double)*ARRAY_SIZE);

	//inital the h_a, h_b
	for(int i = 0;i<ARRAY_SIZE2;i++){
		h_a[i] = double(i);
	}
	for(int i = 0;i<ARRAY_SIZE;i++){
		h_b[i] = double(i);
	}
	for(int i = 0;i<ARRAY_SIZE;i++){
		h_c[i] = double(0);
	}


	////print out test
	//printf("\nThe vector A is:\n");
	//for(int i=0;i<ARRAY_SIZE2;i++){
	//	printf("%f", h_a[i]);
	//	printf(((i%4)!=3)? "\t" : "\n");
	//}

	//printf("\nThe Matrix X is:\n");
	//for(int i=0;i<ARRAY_SIZE;i++){
	//	printf("%f", h_b[i]);
	//	printf(((i%4)!=3)? "\t" : "\n");
	//}


    //// Add vectors in parallel.
    //cudaError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    //if (cudaStatus != cudaSuccess) {
    //    fprintf(stderr, "addWithCuda failed!");
    //    return 1;
    //}

    //printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
    //    c[0], c[1], c[2], c[3], c[4]);

    //// cudaDeviceReset must be called before exiting in order for profiling and
    //// tracing tools such as Nsight and Visual Profiler to show complete traces.
    //cudaStatus = cudaDeviceReset();
    //if (cudaStatus != cudaSuccess) {
    //    fprintf(stderr, "cudaDeviceReset failed!");
    //    return 1;
    //}

	//allocate GPU memory
	hipMalloc((void**)&d_a, sizeof(double)*ARRAY_SIZE2);
    hipMalloc((void**)&d_b, sizeof(double)*ARRAY_SIZE);
    hipMalloc((void**)&d_c, sizeof(double)*ARRAY_SIZE);

	//transfer the array from Host to device(CPU->GPU)
	hipMemcpy(d_a, h_a, sizeof(double)*ARRAY_SIZE2, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(double)*ARRAY_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(d_c, h_c, sizeof(double)*ARRAY_SIZE, hipMemcpyHostToDevice);

	//Run kernel function calculate the matrix-vector multiplication
	printf("\n\nRunning Kernel...\n\n");
    //MVKernel_gm<<<1,256>>>(d_a, d_b, d_c, ARRAY_SIZE);//ARRAY_SIZE/256+1, 256
	addKernel<<<1, ARRAY_SIZE>>>(d_a,d_b,d_c,ARRAY_SIZE);

	//transfer the array from Device to Host(GPU->CPU)
	//cudaMemcpy(h_out, d_out, ARRAY_BYTES, cudaMemcpyDeviceToHost);
	hipMemcpy(h_c, d_c, sizeof(double)*ARRAY_SIZE, hipMemcpyDeviceToHost);

	//print out the result array
	for(int i = 0; i<ARRAY_SIZE;i++){
		printf("%f\n", h_c[i]);
		//printf(((i%4)!=3)? "\t" : "\n");
	}

	//free GPU memory allocation
	hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

	//free Host memory allocation
	free(h_a);
	free(h_b);
	free(h_c);

	system("pause");

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
//cudaError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    cudaError_t cudaStatus;
//
//    // Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = cudaSetDevice(0);
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = cudaMalloc((void**)&dev_c, size * sizeof(int));
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = cudaMalloc((void**)&dev_a, size * sizeof(int));
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = cudaMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = cudaMemcpy(dev_a, a, size * sizeof(int), cudaMemcpyHostToDevice);
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMemcpy failed!");
//        goto Error;
//    }
//
//    cudaStatus = cudaMemcpy(dev_b, b, size * sizeof(int), cudaMemcpyHostToDevice);
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMemcpy failed!");
//        goto Error;
//    }
//
//    // Launch a kernel on the GPU with one thread for each element.
//    //addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = cudaGetLastError();
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", cudaGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // cudaDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = cudaDeviceSynchronize();
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = cudaMemcpy(c, dev_c, size * sizeof(int), cudaMemcpyDeviceToHost);
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    cudaFree(dev_c);
//    cudaFree(dev_a);
//    cudaFree(dev_b);
//
//
//    
//    return cudaStatus;
//}
